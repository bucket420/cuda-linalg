#include "hip/hip_runtime.h"
#include <stdio.h>
#include <random>
#include <time.h>
#include "LinearAlgebra.h"

float dot(Vector *a, Vector *b) {
    float sum = 0.0f;
    for (int i = 0; i < a->size; i++) {
        sum += a->data[i] * b->data[i];
    }
    return sum;
}

__global__ void dotKernel(const float *a, const float *b, float *c, int size) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < size; idx += blockDim.x * gridDim.x) {
        atomicAdd(c, a[idx] * b[idx]);
    }
}

float dotCUDA(Vector *a, Vector *b, int blockSize, int numBlocks) {
    float c = 0.0f;
    float *da, *db, *dc;
    hipMalloc(&da, a->size * sizeof(float));
    hipMalloc(&db, b->size * sizeof(float));
    hipMalloc(&dc, sizeof(float));
    hipMemcpy(da, a->data, a->size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b->data, b->size * sizeof(float), hipMemcpyHostToDevice);
    dotKernel<<<numBlocks, blockSize>>>(da, db, dc, a->size);
    hipMemcpy(&c, dc, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return c;
}